#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include <iostream>

__global__ void kernel( void ) {
}
int main( void ) {
kernel<<<1,1>>>();

int deviceCount;
hipGetDeviceCount(&deviceCount);
printf(" Device Count %i \n",deviceCount);

int device;
for (device = 0; device < deviceCount; ++device) {

hipDeviceProp_t prop;
hipGetDeviceProperties(&prop,device);
printf("Name : %s\n", prop.name);
printf("Device %d has compute capability %d.%d.\n",
           device, prop.major, prop.minor);
printf("totalGlobalMem : %u MB \n" , prop.totalGlobalMem / (1024 * 1024));
printf("sharedMemPerBlock : %u KB \n" , prop.sharedMemPerBlock / 1024 );
printf("regsPerBlock:%d \n", prop.regsPerBlock);
printf("warpSize : %d \n" , prop.warpSize);
printf("memPitch : %u \n", prop.memPitch);
printf("maxThreadPerBlock %d \n" , prop.maxThreadsPerBlock ) ;
printf("maxThreadsDim:x %d, y %d, z %d\n",prop.maxThreadsDim[0],prop.maxThreadsDim[1] , prop.maxThr$
printf("maxGridSize:x %d, y %d, z%d\n", prop.maxGridSize[0],prop.maxGridSize[0] , prop.maxGridSize[$
printf("deviceOverlap:%d \n", prop.deviceOverlap);
printf("totalConstMem:%u\n" , prop.totalConstMem);
printf("major:%d\n",prop.major);
printf("minor:%d\n",prop.minor);
printf("clockRate:%d\n",prop.clockRate);
printf("textureAlignment:%u\n",prop.textureAlignment);
if ( prop.major >= 1 ) {
break;
}
}
return 0;
}