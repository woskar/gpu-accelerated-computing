#include "hip/hip_runtime.h"
// Simplified Histogram Program

#include <iostream>

#define SIZE (100*1024*1024)

__global__ void histo_kernel( unsigned char *buffer,
                              long size,
                              unsigned int *histo ) {

int i = threadIdx.x + blockIdx.x * blockDim.x ;
int offset = blockDim.x * gridDim.x ;

// atomic add on global memory

while  (i < size) {
atomicAdd( &(histo[buffer[i]]), 1 );
      i += offset;
}
}

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <../book.h>

int main( void ) {

unsigned char *buffer = (unsigned char*)big_random_block( SIZE );
unsigned char *dev_buffer;
unsigned int *dev_histo;
unsigned int histo[256];

hipEvent_t	 start, stop;
hipEventCreate( &start );
hipEventCreate( &stop );
hipEventRecord( start,0 );

hipMalloc( (void**)&dev_buffer, SIZE );
hipMalloc( (void**)&dev_histo, 256 * sizeof( long ) );

hipMemcpy( dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);
hipMemset( dev_histo, 0,  256 * sizeof( long ) );

hipDeviceProp_t prop;

hipGetDeviceProperties( &prop, 0 );
int blocks = prop.multiProcessorCount;

// launch histo() kernel with N parallel blocks
histo_kernel<<< blocks*2,256 >>>( dev_buffer, SIZE, dev_histo);
// copy device result back to host copy of d
hipMemcpy( histo, dev_histo, 256*sizeof( int ) , hipMemcpyDeviceToHost);


hipEventRecord( stop,0 );
hipEventSynchronize( stop );

float elapsedTime;

hipEventElapsedTime( &elapsedTime, start, stop );

printf(" Time for histogram generation:  %3.1f ms\n", elapsedTime );

long histoCount = 0;
for (int i=0; i<256; i++) {
         histoCount += histo[i];
}
printf(" Histogram Sum: %1d\n%i", histoCount,histo );

hipEventDestroy( start );
hipEventDestroy( stop );


free( buffer );
hipFree( dev_histo);
hipFree( dev_buffer);

return 0;
}
