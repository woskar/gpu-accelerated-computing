// Histogram Code as in the CUDA Book


#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE (100*1024*1024)

const int threadsPerBlock = 256;

__global__ void histo_kernel( unsigned char *buffer,
                              long size,
                              unsigned int *histo ) {

// Shared memory for adding up histogram on each block
// define a shared vector in the shared memory
__shared__ unsigned int temp[threadsPerBlock];

temp[threadIdx.x] = 0;
__syncthreads(); // to make sure everything is initialized

// atomic add on shared memory

int i = threadIdx.x + blockIdx.x * blockDim.x ;
int offset = blockDim.x * gridDim.x ;

while (i < size) {
        atomicAdd ( &temp[buffer[i]], 1);
      i += offset;
}

__syncthreads();

// atomic add on global memory

atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
/* #include <book.h>   */

void* big_random_block( int size ) {
            unsigned char *data = (unsigned char*)malloc( size );
/* initialize random seed: */
srand ( time(NULL) );

for (int i=0; i<size; i++)
                data[i] = rand();

    return data;
}

int main( void ) {

unsigned char *buffer = (unsigned char*)big_random_block( SIZE );
unsigned char *dev_buffer;
unsigned int *dev_histo;
unsigned int histo[256];

hipEvent_t	 start, stop;
hipEventCreate( &start );
hipEventCreate( &stop );
hipEventRecord( start,0 );

hipMalloc( (void**)&dev_buffer, SIZE );
hipMalloc( (void**)&dev_histo, 256 * sizeof( long ) );

hipMemcpy( dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);
hipMemset( dev_histo, 0,  256 * sizeof( long ) );

hipDeviceProp_t prop;

hipGetDeviceProperties( &prop, 0 );
int blocks = prop.multiProcessorCount;

// launch histo() kernel with N parallel blocks
histo_kernel<<< blocks*2,256 >>>( dev_buffer, SIZE, dev_histo);
// copy device result back to host copy of d
hipMemcpy( histo, dev_histo, 256*sizeof( int ) , hipMemcpyDeviceToHost);


hipEventRecord( stop,0 );
hipEventSynchronize( stop );

float elapsedTime;

hipEventElapsedTime( &elapsedTime, start, stop );

printf(" Time for histogram generation:  %3.1f ms\n", elapsedTime );

long histoCount = 0;
for (int i=0; i<256; i++) {
         histoCount += histo[i];
}
printf(" Histogram Sum: %1d\n", histoCount );

for(int i=0;i<256;printf("histo %i ",histo[i]), i+=256/8 );


hipEventDestroy( start );
hipEventDestroy( stop );

free( buffer );
hipFree( dev_histo);
hipFree( dev_buffer);


return 0;
}
