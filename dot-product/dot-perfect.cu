// dot-perfect is our best version of a dot product of vectors
// fat threads: more than one vector elemnent per thread
// tree-based: reduction of scalar product
// summation of c[i] on the host

const int N = 2048 * 2048;

const int threadsPerBlock = 1024;

__global__ void dot( int *a, int *b, int *c ) {

__shared__ int temp[threadsPerBlock];

int index = threadIdx.x + blockIdx.x *  blockDim.x;
int tempindex = threadIdx.x;

int tempthread = 0;

while (index < N) {
      tempthread += a[index] * b[index];
      index += blockDim.x * gridDim.x;
}

// set the temp value

temp[tempindex] = tempthread;

__syncthreads();

// now reduction, need threadsPerBlock to be power of 2

int i = blockDim.x/2;

while (i != 0) {
    if(tempindex < i)
       temp[tempindex] += temp[tempindex + i];
    __syncthreads();
    i /= 2;
}

if (tempindex == 0)
    c[blockIdx.x] = temp[0];
}


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
/*  #include <book.h> */

#define imin(a,b) (a<b?a:b)

int main( void ) {

int *a, *b,  c, *partial_c ; // host copies of a,b,c,partial_c
int *dev_a, *dev_b, *dev_partial_c; // device copies of a, b, partial_c
int size = N *sizeof( int); // we need space for N integers

hipDeviceProp_t prop;

hipGetDeviceProperties( &prop, 0 );
int blocks = prop.multiProcessorCount;

const int blocksPerGrid =
       imin( blocks, (N+threadsPerBlock-1) / threadsPerBlock );

// allocate device copies of a, b, c
hipMalloc( (void**)&dev_a, size );
hipMalloc( (void**)&dev_b, size );
hipMalloc( (void**)&dev_partial_c, blocksPerGrid*sizeof(int) );

a = (int*)malloc( size );
b = (int*)malloc( size );
partial_c = (int*)malloc( blocksPerGrid*sizeof(int) );

for (int i=0; i<N; i++)
{ a[i] = 1;
};

for (int i=0; i<N; i++)
{ b[i] = 1;
};

// copy inputs to device
hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice);
hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice);


// launch dot() kernel with N parallel blocks
dot<<< blocksPerGrid, threadsPerBlock >>>( dev_a, dev_b, dev_partial_c);

// copy device result back to host copy of d
hipMemcpy( partial_c, dev_partial_c, blocksPerGrid*sizeof(int) , hipMemcpyDeviceToHost);

// finish on the CPU side
c = 0;
for (int i=0; i<blocksPerGrid; i++) {
    c += partial_c[i];
}

printf("a %i b %i ; d %i; \n ",a[0],b[0],c);

free( a ); free( b ); free (partial_c );
hipFree( dev_a);
hipFree( dev_b);
hipFree( dev_partial_c);

return 0;
}
