
#include <hip/hip_runtime.h>
#include <iostream>

#define N (32768)
#define THREADS_PER_BLOCK 1024

__global__ void dot( int *a, int *b, int *d ) {

// Shared memory for results of multiplication

__shared__ int temp[THREADS_PER_BLOCK];
__shared__ int sum;

int index = threadIdx.x + blockIdx.x*blockDim.x ;

temp[threadIdx.x] = a[index] * b[index];
__syncthreads();

// Thread 0 sums the pairwiseproducts
if( 0 == threadIdx.x ) {
sum = 0;
for( int i = 0; i < THREADS_PER_BLOCK ; i++ )
{
sum += temp[i];
};

atomicAdd ( d , sum );
   }
}


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

int main( void ) {

int *a, *b, *d ; // host copies of a,b,c
int *dev_a, *dev_b, *dev_d; // device copies of a, b, c
int size = N *sizeof( int); // we need space for N integers

// allocate device copies of a, b, c
hipMalloc( (void**)&dev_a, size );
hipMalloc( (void**)&dev_b, size );
hipMalloc( (void**)&dev_d, sizeof(int) );

a = (int*)malloc( size );
b = (int*)malloc( size );
d = (int*)malloc( sizeof(int) );

for (int i=0; i<N; i++)
{ a[i] = 1;
};

for (int i=0; i<N; i++)
{ b[i] = 1;
};

*d = 0;

// copy inputs to device
hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice);
hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice);
hipMemcpy( dev_d, d, sizeof(int), hipMemcpyHostToDevice);
// launch dot() kernel with N parallel blocks
dot<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( dev_a, dev_b, dev_d);
// copy device result back to host copy of d
hipMemcpy( d, dev_d, sizeof(int) , hipMemcpyDeviceToHost);

printf("a  %i b  %i ; d  %i; \n ",a[0],b[0],*d);

free( a ); free( b ); free (d );
hipFree( dev_a);
hipFree( dev_b);
hipFree( dev_d);

return 0;
}
