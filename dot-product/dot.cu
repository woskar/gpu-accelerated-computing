// Dot product


#include <hip/hip_runtime.h>
#include <iostream>

#define N 1024

__global__ void dot( int*a, int*b, int*c ) {

__shared__ int temp[N];

temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];

__syncthreads();

// Thread 0 sums the pairwiseproducts
if( 0 == threadIdx.x ) {
int sum = 0;
for( int i = N-1; i >= 0 ; i-- )
sum += temp[i];
c[0] = sum;
   }
}

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

int main( void ) {

int *a, *b, *c ; // host copies of a,b,c
int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
int size = N *sizeof( int); // we need space for N integers

// allocate device copies of a, b, c
hipMalloc( (void**)&dev_a, size );
hipMalloc( (void**)&dev_b, size );
hipMalloc( (void**)&dev_c, size );

a = (int*)malloc( size );
b = (int*)malloc( size );
c = (int*)malloc( size );

for (int i=0; i<N; i++)
{ a[i] = 1;
};

for (int i=0; i<N; i++)
{ b[i] = 1;
};

// copy inputs to device
hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice);
hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice);
// launch dot() kernel with N parallel blocks
dot<<< 1,N >>>( dev_a, dev_b, dev_c);
// copy device result back to host copy of c
hipMemcpy( c, dev_c, size , hipMemcpyDeviceToHost);

printf("a %i b %i ; c %i; \n ",a[0],b[0],c[0]);

free( a ); free( b ); free( c );
hipFree( dev_a);
hipFree( dev_b);
hipFree( dev_c);


return 0;
}