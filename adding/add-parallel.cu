
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add( int*a, int*b, int*c ) {
c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define N 256
int main( void ) {

int *a, *b, *c; // host copies of a,b,c
int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
int size = N *sizeof( int); // we need space for N integers

// allocate device copies of a, b, c
hipMalloc( (void**)&dev_a, size );
hipMalloc( (void**)&dev_b, size );
hipMalloc( (void**)&dev_c, size );

a = (int*)malloc( size );
b = (int*)malloc( size );
c = (int*)malloc( size );

/* initialize random seed: */
srand ( time(NULL) );

for (int i=0; i<N; i+=N/8 )
{ a[i] = rand() %100 + 1;
printf("a %i ",a[i]);
}
printf(" end of a \n\n");


for (int i=0; i<N; i+=N/8 )
{ b[i] = rand() %100 + 1;
printf("b %i ",b[i]);
}
printf(" end of b \n\n");

// copy inputs to device
hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice);
hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice);

// launch add() kernel with N parallel blocks
add<<< N, 1 >>>( dev_a, dev_b, dev_c);

// copy device result back to host copy of c
hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost);

for(int i=0;i<N;printf("c %i ",c[i]), i+=N/8 );

free( a ); free( b ); free( c );
hipFree( dev_a);
hipFree( dev_b);
hipFree( dev_c);

return 0;
}