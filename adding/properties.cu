#include "hip/hip_runtime.h"
#include <iostream>

__global__ void add( int*a, int*b, int*c ) {
c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

int rand ( void );

#define N 512
int main( void ) {
hipDeviceProp_t prop;

int *a, *b, *c; // host copies of a,b,c
int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
int size = N *sizeof( int); // we need space for N integers

  int deviceCount;
    hipGetDeviceCount( &deviceCount ) ;
    printf(" Device Count %i \n",deviceCount);
int i = 0;
for ( i = 0 ; i < deviceCount ; i++) {
hipGetDeviceProperties(&prop,i);
printf("Name : %s\n", prop.name);
printf("totalGlobalMem : %u MB \n" , prop.totalGlobalMem / (1024 * 1024));
printf("sharedMemPerBlock : %u KB \n" , prop.sharedMemPerBlock / 1024 );
printf("regsPerBlock:%d \n", prop.regsPerBlock);
printf("warpSize : %d \n" , prop.warpSize);
printf("memPitch : %u \n", prop.memPitch);
printf("maxThreadPerBlock %d \n" , prop.maxThreadsPerBlock ) ;
printf("maxThreadsDim:x %d, y %d, z %d\n",prop.maxThreadsDim[0],prop.maxThreadsDim[1] , prop.maxThre$
printf("maxGridSize:x %d, y %d, z%d\n", prop.maxGridSize[0],prop.maxGridSize[0] , prop.maxGridSize[1$
printf("deviceOverlap:%d \n", prop.deviceOverlap);
printf("totalConstMem:%u\n" , prop.totalConstMem);
printf("major:%d\n",prop.major);
printf("minor:%d\n",prop.minor);
printf("clockRate:%d\n",prop.clockRate);
printf("textureAlignment:%u\n",prop.textureAlignment);
if ( prop.major >= 1 ) {
break;
}
}

// allocate device copies of a, b, c
hipMalloc( (void**)&dev_a, size );
hipMalloc( (void**)&dev_b, size );
hipMalloc( (void**)&dev_c, size );

a = (int*)malloc( size );
b = (int*)malloc( size );
c = (int*)malloc( size );

for (int i=0; i<N; i++)
{ a[i] = rand() %100 + 1;
printf("a %i ",a[i]);
}
printf(" end of a \n\n");


for (int i=0; i<N; i++)
{ b[i] = rand() %100 + 1;
printf("b %i ",b[i]);
}
printf(" end of b \n\n");

// copy inputs to device
hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice);
hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice);
// launch add() kernel with N parallel blocks
add<<< N, 1 >>>( dev_a, dev_b, dev_c);
// copy device result back to host copy of c
hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost);
free( a ); free( b ); free( c );
hipFree( dev_a);
hipFree( dev_b);
hipFree( dev_c);
for(int i=0;i<N;printf("c %i ",c[i]), i++);

return 0;
}

