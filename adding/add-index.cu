
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add( int*a, int*b, int*c ) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
c[index] = a[index] + b[index];
}

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define N (1024*1024*16)
#define THREADS_PER_BLOCK 1024
int main( void ) {

int *a, *b, *c; // host copies of a,b,c
int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
int size = N *sizeof( int); // we need space for N integers

// allocate device copies of a, b, c
hipMalloc( (void**)&dev_a, size );
hipMalloc( (void**)&dev_b, size );
hipMalloc( (void**)&dev_c, size );

a = (int*)malloc( size );
b = (int*)malloc( size );
c = (int*)malloc( size );

/* initialize random seed: */
srand ( time(NULL) );

for (int i=0; i<N; i ++)
{ a[i] = rand() %100 + 1;
  b[i] = rand() %100 + 1;
};

for (int i=0; i<N; i +=N/100)
{
printf("a %i; ",a[i]);
}
printf(" end of a \n\n");

for (int i=0; i<N; i +=N/100)
{
printf("b %i; ",b[i]);
}
printf(" end of b \n\n");

// copy inputs to device
hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice);
hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice);

// launch add() kernel with N parallel blocks
add<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( dev_a, dev_b, dev_c);

// copy device result back to host copy of c
hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost);

for (int i=0; i<N; i +=N/100)
{
printf("c %i; ",c[i]);
}
printf(" end of c \n");

free( a ); free( b ); free( c );
hipFree( dev_a);
hipFree( dev_b);
hipFree( dev_c);


return 0;
}